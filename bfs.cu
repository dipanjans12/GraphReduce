/******************************************************************************
Copyright 2013 Royal Caliber LLC. (http://www.royal-caliber.com)

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
******************************************************************************/

//BFS using vertexAPI2

#include "util.cuh"
#include "graphio.h"
#include "refgas.h"
#include "gpugas.h"


//nvcc doesn't like the __device__ variable to be a static member inside BFS
//so these are both outside.
int g_iterationCount;
__device__ __constant__ int g_iterationCountGPU;


struct BFS
{
  struct VertexData
  {
    int depth;
  };

  struct EdgeData {}; //nothing

  typedef int GatherResult;
  static const int gatherZero = INT_MAX - 1;

  __host__ __device__
  static int gatherReduce(const int& left, const int& right)
  {
    return 0; //do nothing
  }


  __host__ __device__
  static int gatherMap(
    const VertexData* dst, const VertexData *src, const EdgeData* edge)
  {
    return 0; //do nothing
  }


  __host__ __device__
  static bool apply(VertexData* vert, int dist)
  {
    if( vert->depth == -1 )
    {
      #ifdef __CUDA_ARCH__
        vert->depth = g_iterationCountGPU;
      #else
        vert->depth = g_iterationCount;
      #endif
      return true;
    }
    return false;
  }


  __host__ __device__
  static void scatter(
    const VertexData* src, const VertexData *dst, EdgeData* edge)
  {
    //nothing
  }
};


template<bool GPU>
void setIterationCount(int v)
{
  if( GPU )
    hipMemcpyToSymbol(HIP_SYMBOL(g_iterationCountGPU), &v, sizeof(v));
  else
    g_iterationCount = v;
}


template<typename Engine, bool GPU>
float run(int nVertices, BFS::VertexData* vertexData, int nEdges
  , const int *srcs, const int *dsts, int sourceVertex)
{
  Engine engine;
  int iteration;

  GpuTimer gpu_timer;
  float elapsed = 0.0f;

  // average elapsed time of 10 runs
  for (int itr = 0; itr < 1; ++itr)
  {
    // reset the graph
    for(int i = 0; i < nVertices; ++i) vertexData[i].depth = -1;
    engine.setGraph(nVertices, vertexData, nEdges, 0, &srcs[0], &dsts[0]);
    engine.setActive(sourceVertex, sourceVertex+1);
    iteration = 0;
    setIterationCount<GPU>(iteration);

    gpu_timer.Start();

    while( engine.countActive() )
    {
      //run apply without gather
      engine.gather(false);
      engine.apply();
      engine.scatterActivate(false);
      engine.nextIter();
      setIterationCount<GPU>(++iteration);
    }
    engine.getResults();

    gpu_timer.Stop();
    elapsed += gpu_timer.ElapsedMillis();
  }

  elapsed /= 1;
  // printf("Took %f ms\n", elapsed);
  printf("search depth (number of iterations): %d\n", iteration);
  return elapsed;
}


void outputDepths(int nVertices, BFS::VertexData* vertexData, FILE *f = stdout)
{
  for( int i = 0; i < nVertices; ++i )
    fprintf(f, "%d %d\n", i, vertexData[i].depth);
}


int main(int argc, char** argv)
{
  char *inputFilename;
  char *outputFilename = 0;
  int sourceVertex;
  bool runTest;
  bool dumpResults;
  bool useMaxOutDegreeStart;
  if(!parseCmdLineSimple(argc, argv, "si-t-d-m|s", &inputFilename, &sourceVertex
    , &runTest, &dumpResults, &useMaxOutDegreeStart, &outputFilename) )
  {
    printf("Usage: bfs [-t] [-d] [-m] inputfile source [outputFilename]\n");
    exit(1);
  }

  //load the graph
  int nVertices;
  std::vector<int> srcs;
  std::vector<int> dsts;
  loadGraph(inputFilename, nVertices, srcs, dsts);

  //initialize vertex data
  std::vector<BFS::VertexData> vertexData(nVertices);
  for( int i = 0; i < nVertices; ++i )
    vertexData[i].depth = -1;

/*
  useMaxOutDegreeStart is a boolean which indicates whether to use 
  the start vertex to be the default i.e. 1 or the vertex with the
  maximum out degree.
*/
  if( useMaxOutDegreeStart )
  {
    //convert to CSR layout to find source vertex
    std::vector<int> srcOffsets(nVertices + 1);
    std::vector<int> csrSrcs(srcs.size());
    edgeListToCSR<int>(
      nVertices, srcs.size(), &srcs[0], &dsts[0], &srcOffsets[0], 0, 0);
    int maxDegree = -1;
    sourceVertex = -1;
    for(int i = 0; i < nVertices; ++i)
    {
      int outDegree = srcOffsets[i + 1] - srcOffsets[i];
      if( outDegree > maxDegree )
      {
        maxDegree    = outDegree;
        sourceVertex = i;
      }
    }
    printf(
      "using vertex %d with degree %d as source\n", sourceVertex, maxDegree);
  }

  std::vector<BFS::VertexData> refVertexData;
  if( runTest )
  {
    refVertexData = vertexData;
    float elapsed = run<GASEngineRef<BFS>, false>(nVertices
      , &refVertexData[0], (int)srcs.size(), &srcs[0], &dsts[0], sourceVertex);
    if( dumpResults )
    {
      printf("Reference:\n");
      outputDepths(nVertices, &refVertexData[0]);
    }
  }

/*
  The following line calls the run() function which is the main
  function that executes the graph algorithm.
  true indicates that the code is to be run on GPU.
  false indicates that the code is to be run on CPU.
*/
  float elapsed = run<GASEngineGPU<BFS>, true>(nVertices, &vertexData[0]
    , (int) srcs.size(), &srcs[0], &dsts[0], sourceVertex);

  // compute stats
  int nodes_visited = 0;
  int edges_visited = 0;
  std::vector<int> srcOffsets(nVertices + 1);
  std::vector<int> csrSrcs(srcs.size());
  edgeListToCSR<int>(
    nVertices, srcs.size(), &srcs[0], &dsts[0], &srcOffsets[0], 0, 0);

  for (int itr = 0; itr < nVertices; ++itr)
  {
    if (vertexData[itr].depth > -1)
    {
      nodes_visited += 1;
      edges_visited += srcOffsets.at(itr+1) - srcOffsets.at(itr);
    }
  }

  printf("nodes visited: %d edges visited: %d\n", nodes_visited, edges_visited);
  float m_teps = (float) edges_visited / (elapsed * 1000);
  printf("elapsed: %.4f ms, MTEPS: %.4f MiEdges/s\n", elapsed, m_teps);

  if( dumpResults )
  {
    printf("GPU:\n");
    outputDepths(nVertices, &vertexData[0]);
  }

  if( runTest )
  {
    bool diff = false;
    for( int i = 0; i < nVertices; ++i )
    {
      if( vertexData[i].depth != refVertexData[i].depth )
      {
        printf("%d %d %d\n", i, refVertexData[i].depth, vertexData[i].depth);
        diff = true;
      }
    }
    if( diff )
      return 1;
    else
      printf("No differences found\n");
  }

  if( outputFilename )
  {
    printf("writing results to %s\n", outputFilename);
    FILE* f = fopen(outputFilename, "w");
    outputDepths(nVertices, &vertexData[0], f);
    fclose(f);
  }

  free(inputFilename);
  free(outputFilename);

  return 0;
}
