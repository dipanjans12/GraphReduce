#include "hip/hip_runtime.h"
/******************************************************************************
Copyright 2013 Royal Caliber LLC. (http://www.royal-caliber.com)

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
******************************************************************************/

//Single source shortest paths using vertexAPI2

#include "util.cuh"
#include "graphio.h"
#include "refgas.h"
#include "gpugas.h"
#include <climits>

struct SSSP
{
  //making these typedefs rather than singleton structs
  typedef int VertexData;
  typedef int EdgeData;

  typedef int GatherResult;
  static const int maxLength = 100000;
  static const int gatherZero = INT_MAX - maxLength;


  __host__ __device__
  static int gatherReduce(const int& left, const int& right)
  {
    return min(left, right);
  }


  __host__ __device__
  static int gatherMap(
    const VertexData* dstDist, const VertexData *srcDist, const EdgeData* edgeLen)
  {
    return *srcDist + *edgeLen;
  }


  __host__ __device__
  static bool apply(VertexData* curDist, GatherResult dist)
  {
    bool changed = dist < *curDist;
    *curDist = min(*curDist, dist);
    return changed;
  }


  __host__ __device__
  static void scatter(
    const VertexData* src, const VertexData *dst, EdgeData* edge)
  {
    //nothing
  }
};


template<typename Engine>
float run(int srcVertex, int nVertices, SSSP::VertexData* vertexData, int nEdges
  , SSSP::EdgeData* edgeData, const int* srcs, const int* dsts)
{
  Engine engine;

  GpuTimer gpu_timer;
  float elapsed = 0.0f;
  int iteration = 0;

  // average elapsed time of 10 runs
  for (int itr = 0; itr < 10; ++itr)
  {
    // reset the graph
    for(int i = 0; i < nVertices; ++i) vertexData[i] = SSSP::gatherZero;
    vertexData[srcVertex] = 0;
    engine.setGraph(nVertices, vertexData, nEdges, edgeData, srcs, dsts);

    //TODO, setting all vertices to active for first step works, but it would
    //be faster to instead set to neighbors of starting vertex
    engine.setActive(0, nVertices);

    gpu_timer.Start();

    while (engine.countActive())
    {
      engine.gather();
      engine.apply();
      engine.scatterActivate();
      engine.nextIter();
      ++iteration;
    }

    engine.getResults();

    gpu_timer.Stop();
    elapsed += gpu_timer.ElapsedMillis();
  }

  elapsed /= 10;
  printf("number of iterations: %d\n", iteration);
  return elapsed;
}


void outputDists(int nVertices, int* dists, FILE* f = stdout)
{
  for (int i = 0; i < nVertices; ++i)
    fprintf(f, "%d %d\n", i, dists[i]);
}


int main(int argc, char** argv)
{
  char *inputFilename;
  char *outputFilename = 0;
  int sourceVertex;
  bool runTest;
  bool dumpResults;
  bool useMaxOutDegreeStart;
  if(!parseCmdLineSimple(argc, argv, "si-t-d-m|s", &inputFilename, &sourceVertex
    , &runTest, &dumpResults, &useMaxOutDegreeStart, &outputFilename) )
  {
    printf("Usage: sssp [-t] [-d] [-m] inputfile source [outputfile]\n");
    exit(1);
  }

  //load the graph
  int nVertices;
  std::vector<int> srcs;
  std::vector<int> dsts;
  std::vector<int> edgeData;
  loadGraph(inputFilename, nVertices, srcs, dsts, &edgeData);
  if( edgeData.size() == 0 )
  {
    printf("No edge data available in input file\n");
    exit(1);
  }

  if( useMaxOutDegreeStart )
  {
    //convert to CSR layout to find source vertex
    std::vector<int> srcOffsets(nVertices + 1);
    std::vector<int> csrSrcs(srcs.size());
    edgeListToCSR<int>(
      nVertices, srcs.size(), &srcs[0], &dsts[0], &srcOffsets[0], 0, 0);
    int maxDegree = -1;
    sourceVertex = -1;
    for(int i = 0; i < nVertices; ++i)
    {
      int outDegree = srcOffsets[i + 1] - srcOffsets[i];
      if( outDegree > maxDegree )
      {
        maxDegree    = outDegree;
        sourceVertex = i;
      }
    }
    printf(
      "using vertex %d with degree %d as source\n", sourceVertex, maxDegree);
  }


  //initialize vertex data
  std::vector<int> vertexData(nVertices);
  for( int i = 0; i < nVertices; ++i )
    vertexData[i] = SSSP::gatherZero;
  vertexData[sourceVertex] = 0;

  std::vector<int> refVertexData;
  if( runTest )
  {
    printf("Running reference calculation\n");
    refVertexData = vertexData;
    float elapsed = run< GASEngineRef<SSSP> >(sourceVertex, nVertices
      , &refVertexData[0], (int)srcs.size(), &edgeData[0], &srcs[0], &dsts[0]);
    if( dumpResults )
    {
      printf("Reference:\n");
      outputDists(nVertices, &refVertexData[0]);
    }
  }

  float elapsed = run< GASEngineGPU<SSSP> >(sourceVertex, nVertices
    , &vertexData[0], (int)srcs.size(), &edgeData[0], &srcs[0], &dsts[0]);

  // compute stats
  long int nodes_visited = 0;
  long int edges_visited = 0;
  std::vector<int> srcOffsets(nVertices + 1);
  std::vector<int> csrSrcs(srcs.size());
  edgeListToCSR<int>(
    nVertices, srcs.size(), &srcs[0], &dsts[0], &srcOffsets[0], 0, 0);

  for (int itr = 0; itr < nVertices; ++itr)
  {
    if (vertexData.at(itr) < SSSP::gatherZero)
    {
      nodes_visited += 1;
      edges_visited += srcOffsets.at(itr+1) - srcOffsets.at(itr);
    }
  }

  printf("nodes visited: %d edges visited: %d\n", nodes_visited, edges_visited);
  float m_teps = (float) edges_visited / (elapsed * 1000);
  printf("elapsed: %.4f ms, MTEPS: %.4f MiEdges/s\n", elapsed, m_teps);

  if( dumpResults )
  {
    printf("GPU:\n");
    outputDists(nVertices, &vertexData[0]);
  }

  if( runTest )
  {
    bool diff = false;
    for( int i = 0; i < nVertices; ++i )
    {
      if( vertexData[i] != refVertexData[i] )
      {
        printf("%d %d %d\n", i, refVertexData[i], vertexData[i]);
        diff = true;
      }
    }
    if( diff )
      return 1;
    else
      printf("No differences found\n");
  }

  if( outputFilename )
  {
    printf("writing results to %s\n", outputFilename);
    FILE* f = fopen(outputFilename, "w");
    outputDists(nVertices, &vertexData[0], f);
    fclose(f);
  }

  free(inputFilename);
  free(outputFilename);

  return 0;
}
