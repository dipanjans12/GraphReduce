#include "hip/hip_runtime.h"
/******************************************************************************
Copyright 2013 Royal Caliber LLC. (http://www.royal-caliber.com)

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
******************************************************************************/


#include "refgas.h"
#include "gpugas.h"
#include "util.cuh"
#include "graphio.h"
#include <vector>
#include <iostream>


//Vertex program for Pagerank
struct PageRank
{
  static const float pageConst = 0.15f;
  static const float tol = 0.01f;

  struct VertexData
  {
    float rank;
    int   numOutEdges;
    friend std::ostream& operator<<(std::ostream &out, const VertexData &data);
  };

  struct EdgeData {};

  typedef float GatherResult;

  static const float gatherZero = 0.0f;

  __host__ __device__
  static float gatherMap(const VertexData* dst, const VertexData* src, const EdgeData* edge)
  {
    //this division is being done too many times right?
    //should just store the normalized value in apply?
    return src->rank / src->numOutEdges;
  }

  __host__ __device__
  static float gatherReduce(const float& left, const float& right)
  {
    return left + right;
  }

  __host__ __device__
  static bool apply(VertexData* vertexData, const float& gatherResult)
  {
    float newRank = pageConst + (1.0f - pageConst) * gatherResult;
    bool ret = fabs(newRank - vertexData->rank) >= tol;
    vertexData->rank = newRank;
    return ret;
  }

  __host__ __device__
  static void scatter(const VertexData* src, const VertexData *dst, EdgeData* edge)
  {
    //nothing
  }
};


void outputRanks(int n, const PageRank::VertexData* vertexData, FILE* f = stdout)
{
  for( int i = 0; i < n; ++i )
  {
    fprintf(f, "%d %f\n", i, vertexData[i].rank);
  }
}


template<typename Engine>
void run(int nVertices, PageRank::VertexData* vertexData, int nEdges
  , const int* srcs, const int* dsts)
{
  for( int i = 0; i < nVertices; ++i )
    vertexData[i].rank = PageRank::pageConst;

  Engine engine;
  engine.setGraph(nVertices, vertexData, nEdges, 0, srcs, dsts);
  //all vertices begin active for pagerank
  engine.setActive(0, nVertices);
  int64_t t0 = currentTime();
  engine.run();
  engine.getResults();
  int64_t t1 = currentTime();
  printf("Took %f ms\n", (t1 - t0)/1000.0f);
}


int main(int argc, char **argv)
{
  char* inputFilename;
  char* outputFilename = 0;
  bool runTest;
  bool dumpResults;
  if( !parseCmdLineSimple(argc, argv, "s-t-d|s"
    , &inputFilename, &runTest, &dumpResults, &outputFilename) )
  {
    printf("Usage: pagerank [-t] [-d] inputfile [outputfile]\n");
    exit(1);
  }

  //load the graph
  int nVertices;
  std::vector<int> srcs;
  std::vector<int> dsts;
  loadGraph(inputFilename, nVertices, srcs, dsts);
  printf("loaded %s with %d vertices and %zd edges\n", inputFilename, nVertices, srcs.size());

  //initialize vertex data
  //convert to CSR to get the count of edges.
  std::vector<int> srcOffsets(nVertices + 1);
  std::vector<int> csrSrcs(srcs.size());
  edgeListToCSR<int>(nVertices, srcs.size(), &srcs[0], &dsts[0], &srcOffsets[0], 0, 0);

  std::vector<PageRank::VertexData> vertexData(nVertices);
  for( int i = 0; i < nVertices; ++i )
    vertexData[i].numOutEdges = srcOffsets[i + 1] - srcOffsets[i];

  std::vector<PageRank::VertexData> refVertexData;
  if( runTest )
  {
    printf("Running reference calculation\n");
    refVertexData = vertexData;
    run< GASEngineRef<PageRank> >(nVertices, &refVertexData[0], (int)srcs.size(), &srcs[0], &dsts[0]);
    if( dumpResults )
    {
      printf("Reference\n");
      outputRanks(nVertices, &refVertexData[0]);
    }
  }

  run< GASEngineGPU<PageRank> >(nVertices, &vertexData[0], (int)srcs.size(), &srcs[0], &dsts[0]);
  if( dumpResults )
  {
    printf("GPU:\n");
    outputRanks(nVertices, &vertexData[0]);
  }

  if( runTest )
  {
    const float tol = 1.0e-6f;
    bool diff = false;
    for( int i = 0; i < nVertices; ++i )
    {
      if( fabs(vertexData[i].rank - refVertexData[i].rank) > tol )
      {
        printf("%d %f %f\n", i, refVertexData[i].rank, vertexData[i].rank);
        diff = true;
      }
    }
    if( diff )
      return 1;
    else
      printf("No differences found\n");
  }

  if( outputFilename )
  {
    FILE* f = fopen(outputFilename, "w");
    printf("writing results to file %s\n", outputFilename);
    outputRanks(nVertices, &vertexData[0], f);
    fclose(f);
  }

  free(inputFilename);
  free(outputFilename);

  return 0;
}

std::ostream& operator<<(std::ostream &out, const PageRank::VertexData &data) {
  out << data.rank;
  return out;
}
